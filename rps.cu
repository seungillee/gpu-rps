#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <iostream>
#include <vector>

#define N 10 // Number of rounds

__global__ void randomChoiceKernel(int *choices, unsigned long seed)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N)
    {
        hiprandState state;
        hiprand_init(seed, idx, 0, &state);
        choices[idx] = hiprand(&state) % 3; // Generate random 0, 1, or 2
    }
}

void determineWinner(const std::vector<int> &gpu1, const std::vector<int> &gpu2)
{
    for (int i = 0; i < N; ++i)
    {
        std::string winner;
        if (gpu1[i] == gpu2[i])
        {
            winner = "Tie";
        }
        else if ((gpu1[i] == 0 && gpu2[i] == 2) ||
                 (gpu1[i] == 1 && gpu2[i] == 0) ||
                 (gpu1[i] == 2 && gpu2[i] == 1))
        {
            winner = "GPU 1 Wins";
        }
        else
        {
            winner = "GPU 2 Wins";
        }
        std::cout << "Round " << i + 1 << ": GPU 1 -> " << gpu1[i]
                  << ", GPU 2 -> " << gpu2[i] << " | " << winner << "\n";
    }
}

int main()
{
    int *d_gpu1, *d_gpu2;
    int h_gpu1[N], h_gpu2[N];

    hipMalloc(&d_gpu1, N * sizeof(int));
    hipMalloc(&d_gpu2, N * sizeof(int));

    randomChoiceKernel<<<1, N>>>(d_gpu1, time(0));     // GPU 1
    randomChoiceKernel<<<1, N>>>(d_gpu2, time(0) + 1); // GPU 2

    hipMemcpy(h_gpu1, d_gpu1, N * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(h_gpu2, d_gpu2, N * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_gpu1);
    hipFree(d_gpu2);

    std::vector<int> gpu1(h_gpu1, h_gpu1 + N);
    std::vector<int> gpu2(h_gpu2, h_gpu2 + N);

    determineWinner(gpu1, gpu2);

    return 0;
}